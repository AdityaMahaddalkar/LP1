
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 512

__global__ void scan(float *input, float *output, int len) {
    __shared__ float data[BLOCK_SIZE];

    // DEBUG
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("Block Number: %d\n", blockIdx.x);
        for (int i = 0; i < BLOCK_SIZE; ++i)
        {
            printf("DATA[%d] = %f\n", i, data[i]);
        }
    }

}

int main(int argc, char ** argv) {
    dim3 block(BLOCK_SIZE, 1, 1);
    dim3 grid(10, 1, 1);
    scan<<<grid,block>>>(NULL, NULL, NULL);
    hipDeviceSynchronize();
    return 0;
}