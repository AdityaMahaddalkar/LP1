
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstdlib>
#include <cmath>
using namespace std;
using namespace std::chrono;

__global__ void reduce(int *g_idata, int *g_odata){
    extern __shared__ int sdata[];

    //each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s = 1;s < blockDim.x; s *= 2){
        if(tid % (2 * s) == 0){
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void compute_difference_between_mean_and_elements(int *difference_array, int *original_array, int mean){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    difference_array[tid] = (original_array[tid] - mean) * (original_array[tid] - mean);
}

void sum_CPU(int *host_input, int *host_output, unsigned int size){
    host_output[0] = 0;
    auto start = high_resolution_clock::now();
    for(int i = 0;i < size;i ++){
        host_output[0] += host_input[i];
    }
    auto stop = high_resolution_clock::now();
    auto time_req = duration_cast<microseconds>(stop - start).count();
    cout << endl << "Time required for CPU : " << time_req << " microseconds "<< endl;
    cout << endl << " Sum from CPU : " << host_output[0] << endl;
}

void compute_sum_cpu(int *cpu_input, int *cpu_output, unsigned int n){
    
    for(unsigned int i = 0;i < n;i ++){
        cpu_output[0] += cpu_input[i];
    }
    
}

int main(){
    
    int maxThreads = 1024;
    
    int *host_input, *host_output, *device_input, *device_output;
    int *cpu_input, *cpu_output;

    int n = 2 << 12;
    size_t size = n * sizeof(int);

    //CPU sum
    cpu_input = (int *)malloc(size);
    cpu_output = (int *)malloc(sizeof(int));
    cpu_output[0] = 0;

    for(unsigned int i = 0;i < n;i ++){
        cpu_input[i] = rand()%10;
    }

    sum_CPU(cpu_input, cpu_output, n);

    host_input = (int *)malloc(size);
    for(int i = 0;i < n;i ++){
        host_input[i] = cpu_input[i];
    }
    
    int blocks = n / maxThreads;
    host_output = (int *)malloc(blocks * sizeof(int));

    const dim3 block_size(maxThreads, 1, 1);
    const dim3 grid_size(blocks, 1, 1);
    
    hipMalloc(&device_input, size);
    hipMalloc(&device_output, blocks * sizeof(int));

    hipMemcpy(device_input, host_input, size, hipMemcpyHostToDevice);

    reduce<<<grid_size, block_size, maxThreads * sizeof(int)>>>(device_input, device_output);

    hipMemcpy(host_output, device_output, blocks * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 1;i < blocks; i++){
        host_output[0] += host_output[i];
    }

    cout << endl << " Sum from GPU : " << *host_output << endl;
    
    int mean = int(host_output[0] / n);
    cout << endl << " Mean of the array : " << mean << endl;

    //Compute array of [(x1-mean)^2, (x2-mean)^2, (x3-mean)^2, ... ]
    int *array_of_difference_between_mean_and_elements_device;
    hipMalloc(&array_of_difference_between_mean_and_elements_device, size);
    compute_difference_between_mean_and_elements<<<grid_size, block_size>>>(array_of_difference_between_mean_and_elements_device, device_input, mean);

    //Compute (x1-mean)^2 + (x2 - mean) ^ 2 + ...
    int *output_array_for_sum_of_difference_between_elements, *output_array_for_sum_of_difference_between_elements_host;
    output_array_for_sum_of_difference_between_elements_host = (int *)malloc(blocks * sizeof(int));
    hipMalloc(&output_array_for_sum_of_difference_between_elements, blocks * sizeof(int));
    reduce<<<grid_size, block_size, maxThreads * sizeof(int)>>>(array_of_difference_between_mean_and_elements_device, output_array_for_sum_of_difference_between_elements);
    hipMemcpy(output_array_for_sum_of_difference_between_elements_host, output_array_for_sum_of_difference_between_elements, blocks * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 1;i < blocks;i ++){
        output_array_for_sum_of_difference_between_elements_host[0] += output_array_for_sum_of_difference_between_elements_host[i];
    }
    
    // Compute variance i.e ((x1 - mean)^2 + (x2 - mean)^2 ...) / n
    output_array_for_sum_of_difference_between_elements_host[0] = output_array_for_sum_of_difference_between_elements_host[0] / n;
    cout << endl << "Variance from GPU : " << output_array_for_sum_of_difference_between_elements_host[0] << endl;

    //Compute square root of (x1 - mean) ^ 2 + (x2 - mean) ^ 2 ...
    output_array_for_sum_of_difference_between_elements_host[0] = sqrt(output_array_for_sum_of_difference_between_elements_host[0]);

    cout << endl << "Standard deviation from  GPU : " << output_array_for_sum_of_difference_between_elements_host[0] << endl;

}